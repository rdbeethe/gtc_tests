#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <inttypes.h>


#define MAX_BLOCK_SIZE 32
#define MAX_WINDOW_SIZE 55
#define MAX_DISP 1000

#define NCHANS 3

#define BLOCK_SIZE 16

// timing utility
struct timespec check_timer(const char* str, struct timespec* ts){
	struct timespec oldtime;
	// copy old time over
	oldtime.tv_nsec = ts->tv_nsec;
	oldtime.tv_sec = ts->tv_sec;
	// update ts
	clock_gettime(CLOCK_REALTIME, ts);
	// print old time
	int diffsec;
	int diffnsec;
	if(str != NULL){
		diffsec =  ts->tv_sec - oldtime.tv_sec;
		diffnsec =  ts->tv_nsec - oldtime.tv_nsec;
		// correct the values if we measured over an integer second break:
		if(diffnsec < 0){
			diffsec--;
			diffnsec += 1000000000;
		}
		printf("%s:%ds %fms\n",str,diffsec,diffnsec/1e6);
	}
	return (struct timespec) {diffsec, diffnsec};
}

// little bitty kernel to initialize blocks of device memory
__global__ void gpu_memset(unsigned char* start, unsigned char value, int length){
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int gx = bx*blockDim.x + tx;
	if(gx < length){
		start[gx] = value;
	}
}

// teeny little helper function
void gpu_perror(char* input){
	printf("%s: %s\n", input, hipGetErrorString(hipGetLastError()));
}

// Device code
__global__ void asw_kernel(unsigned char* global_left, unsigned char* global_right, unsigned char* output, unsigned char* debug,
	int nrows, int ncols, int nchans, int ndisp, int win_size, int win_rad, float s_sigma, float c_sigma)
	{
	extern __shared__ unsigned char ref[]; // contains both left and right image data

	// get the size of the sub-images that we are considering
	// reference window
	int ref_width_bytes = (2*win_rad+blockDim.x)*NCHANS*sizeof(unsigned char);
	// int ref_rows = (2*win_rad+blockDim.y);
	// target window
	int tgt_width_bytes = (ndisp+2*win_rad+blockDim.x)*NCHANS*sizeof(unsigned char);
	// int tgt_rows = (2*win_rad+blockDim.y);

	unsigned char* tgt = (unsigned char*)(&ref[ ref_width_bytes*(2*win_rad+blockDim.y) ]); // tgt image, reference to somwhere of shared allocated memory

	float ref_c_factor;
	float tgt_c_factor;
	float s_factor;
	float ref_c2p_diff;
	float tgt_c2p_diff;
	float ref2tgt_diff;
	// variables for keeping track of the output
	float weight;
	float cost;
	float min_cost;
	unsigned char min_cost_index;
	unsigned char ref_center_pix[3];
	// unsigned char tgt_center_pix[3];
	unsigned char ref_pix[3];
	unsigned char tgt_pix[3];

	int disp;
	int win_x;
	int win_y;
	int dx;
	int tgt_x;

	// get identity of this thread (changing these to #define's)

	#define tx (threadIdx.x)
	#define ty (threadIdx.y)
	#define bx (blockIdx.x + 5)
	#define by (blockIdx.y + 1)
	#define gx (bx*blockDim.x + tx)
	#define gy (by*blockDim.y + ty)

	// setup LUTs // nevermind... right now there are none

	// copy relevant subimages to shared memory
	// TODO: additional boundary checks on this data
	// TODO: better division technique
	// TODO: investigate where syncthreads() needs to be called for best performance
	// starting with reference image: (4 deleted register variables)	
	// int xblocks = (ref_width_bytes / blockDim.x + 1);
	// int yblocks = ((2*win_rad+blockDim.y) / blockDim.y + 1);
	// int xstart = ((bx*blockDim.x - win_rad)*NCHANS);
	// int ystart = (gy - win_rad);
	for(win_x = 0; win_x < (ref_width_bytes / blockDim.x + 1); win_x++){
		// int x_idx = (win_x*blockDim.x + tx);
		// int g_x_idx = (((bx*blockDim.x - win_rad)*NCHANS) + win_x*blockDim.x + tx);
		if((win_x*blockDim.x + tx) < ref_width_bytes){
			for(win_y = 0; win_y < ((2*win_rad+blockDim.y) / blockDim.y + 1); win_y++){
				// int y_idx = (win_y*blockDim.y + ty);
				// int g_y_idx = ((gy - win_rad) + win_y*blockDim.y);
				if((win_y*blockDim.y + ty) < (2*win_rad+blockDim.y)){
					// copy bytes (not pixels) from global_left into reference image
					ref[(win_y*blockDim.y + ty)*ref_width_bytes + (win_x*blockDim.x + tx)] = global_left[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad)*NCHANS) + win_x*blockDim.x + tx)];
					// copy into the debug image (only made to work with a single block of threads)
					// debug[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad)*NCHANS) + win_x*blockDim.x + tx)]  = ref[(win_y*blockDim.y + ty)*ref_width_bytes + (win_x*blockDim.x + tx)];
				}
			}
		}
	}
	// then to the target image: (4 deleted register variables)
	// xblocks = (tgt_width_bytes / blockDim.x + 1);
	// yblocks = ((2*win_rad+blockDim.y) / blockDim.y + 1);
	// xstart = ((bx*blockDim.x - win_rad - ndisp)*NCHANS);
	// ystart = (gy - win_rad);
	for(win_x = 0; win_x < (tgt_width_bytes / blockDim.x + 1); win_x++){
		// int x_idx = (win_x*blockDim.x + tx);
		// int g_x_idx = (((bx*blockDim.x - win_rad - ndisp)*NCHANS) + win_x*blockDim.x + tx);
		if((win_x*blockDim.x + tx) < tgt_width_bytes){
			for(win_y = 0; win_y < ((2*win_rad+blockDim.y) / blockDim.y + 1); win_y++){
				// int y_idx = (win_y*blockDim.y + ty);
				// int g_y_idx = ((gy - win_rad) + win_y*blockDim.y);
				if((win_y*blockDim.y + ty) < (2*win_rad+blockDim.y)){
					// copy bytes (not pixels) from global_left into reference image
					tgt[(win_y*blockDim.y + ty)*tgt_width_bytes + (win_x*blockDim.x + tx)] = global_right[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad - ndisp)*NCHANS) + win_x*blockDim.x + tx)];
					// copy into the debug image (only made to work with a single block of threads)
					// debug[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad - ndisp)*NCHANS) + win_x*blockDim.x + tx)]  = tgt[(win_y*blockDim.y + ty)*tgt_width_bytes + (win_x*blockDim.x + tx)];
				}
			}
		}
	}

	__syncthreads();

	// get a pointer to the ref_center_pix, which is constant for any given thread
	ref_center_pix[0] = ref[(win_rad + ty)*ref_width_bytes + (win_rad + tx)*NCHANS + 0];
	ref_center_pix[1] = ref[(win_rad + ty)*ref_width_bytes + (win_rad + tx)*NCHANS + 1];
	ref_center_pix[2] = ref[(win_rad + ty)*ref_width_bytes + (win_rad + tx)*NCHANS + 2];
	// initialize min_cost to some arbitrarily large value
	min_cost = 1e12;
	// initialize min_cost_index to 0
	min_cost_index = 0;

	// for each value of ndisp	
	for(disp = 0; disp < ndisp; disp++){
		// get a pointer to the tgt_center_pix, which is constant for each disp
		// ... except I get better results by using ref_center_pix to compare to tgt_pix
		// tgt_center_pix[0] = tgt[(win_rad + ty)*tgt_width_bytes + (ndisp + win_rad + tx - disp)*NCHANS + 0];
		// tgt_center_pix[1] = tgt[(win_rad + ty)*tgt_width_bytes + (ndisp + win_rad + tx - disp)*NCHANS + 1];
		// tgt_center_pix[2] = tgt[(win_rad + ty)*tgt_width_bytes + (ndisp + win_rad + tx - disp)*NCHANS + 2];
		// reset weight and cost
		weight = 0;
		cost = 0;
		// in each row in the window:
		for(win_x = 0; win_x < win_size; win_x++){
			// locate the pixel in the ref image (deleted this var)
			dx = win_x + tx;
			// locate the pixel in the tgt image (deleted this var)
			tgt_x = ndisp + win_x + tx - disp;
			// find the window-center to pixel x-distance (deleted this var)
			// int dx = win_x - win_rad;
			// in each column of the window:
			for(win_y = 0; win_y < win_size; win_y++){
				// locate the pixel in the ref image (deleted this var)
				// int ref_y = win_y + ty;
				// find the window-center to pixel y-distance (deleted this var)
				// int dy = win_y - win_rad;
				// get the radius^2 value (deleted this var)
				// float radius_2 = (win_x-win_rad)*(win_x-win_rad) + (win_y-win_rad)*(win_y-win_rad);
				// get the s_factor for this particular window location
				s_factor = __expf(-((win_x-win_rad)*(win_x-win_rad) + (win_y-win_rad)*(win_y-win_rad))/(2.*s_sigma*s_sigma));
				// store tgt and ref pixels in register memory
				ref_pix[0] = ref[(win_y+ty)*ref_width_bytes + (dx)*NCHANS + 0];
				ref_pix[1] = ref[(win_y+ty)*ref_width_bytes + (dx)*NCHANS + 1];
				ref_pix[2] = ref[(win_y+ty)*ref_width_bytes + (dx)*NCHANS + 2];
				tgt_pix[0] = tgt[(win_y+ty)*tgt_width_bytes + (tgt_x)*NCHANS + 0];
				tgt_pix[1] = tgt[(win_y+ty)*tgt_width_bytes + (tgt_x)*NCHANS + 1];
				tgt_pix[2] = tgt[(win_y+ty)*tgt_width_bytes + (tgt_x)*NCHANS + 2];
				// get the center-to-pixel and overall color differences (organized together for IDP)
				ref_c2p_diff = abs(ref_center_pix[0] - ref_pix[0]);
				tgt_c2p_diff = abs(ref_center_pix[0] - tgt_pix[0]);
				ref2tgt_diff = abs(ref_pix[0] - tgt_pix[0]);
				ref_c2p_diff += abs(ref_center_pix[1] - ref_pix[1]);
				tgt_c2p_diff += abs(ref_center_pix[1] - tgt_pix[1]);
				ref2tgt_diff+= abs(ref_pix[1] - tgt_pix[1]);
				ref_c2p_diff += abs(ref_center_pix[2] - ref_pix[2]);
				tgt_c2p_diff += abs(ref_center_pix[2] - tgt_pix[2]);
				ref2tgt_diff+= abs(ref_pix[2] - tgt_pix[2]);
				// get the c_factors
				ref_c_factor = __expf(-ref_c2p_diff*ref_c2p_diff/(2.*c_sigma*c_sigma));
				tgt_c_factor = __expf(-tgt_c2p_diff*tgt_c2p_diff/(2.*c_sigma*c_sigma));
				// calulate the pix_weight (this variable has been done away with to increase ILP)
				// pix_weight = s_factor*ref_c_factor*tgt_c_factor;
				// add in the cost
				cost += s_factor*ref_c_factor*tgt_c_factor*ref2tgt_diff;
				// add in the weight
				weight += s_factor*ref_c_factor*tgt_c_factor;
			}
		}
		// now that the window is done, compare this cost (after normalizing) to min_cost
		if( min_cost > cost / weight){
			min_cost = cost / weight;
			min_cost_index = disp;
		}
		__syncthreads();
	}

	// set the output to the index of min_cost
	output[gy*ncols + gx] = min_cost_index;
}

int asw(cv::Mat im_l, cv::Mat im_r, int ndisp, int s_sigma, int c_sigma){
	// window size and win_rad
	int win_rad = 1.5*s_sigma;
	int win_size = 2*win_rad+1;
	// declare timer
	struct timespec ts;

	// check that images are matching dimensions
	if(im_l.rows != im_r.rows){
		printf("Error: im_l and im_r do not have matching row count\n");
		return 1;
	}
	if(im_l.cols != im_r.cols){
		printf("Error: im_l and im_r do not have matching col count\n");
		return 1;
	}
	if(im_l.channels() != im_r.channels()){
		printf("Error: im_l and im_r do not have matching channel count\n");
		return 1;
	}

	// set easy-access variables for number of rows, cols, and chans
	int nrows = im_l.rows;
	int ncols = im_l.cols;
	int nchans = im_l.channels();
	// initialize the device input arrays
	unsigned char* d_im_l;
	hipMalloc(&d_im_l,nchans*nrows*ncols*sizeof(unsigned char));
	unsigned char* d_im_r;
	hipMalloc(&d_im_r,nchans*nrows*ncols*sizeof(unsigned char));
	// initialize the output data matrix
	unsigned char* out = (unsigned char*)malloc(nrows*ncols*sizeof(unsigned char));
	unsigned char* d_out;
	hipMalloc(&d_out,nrows*ncols*sizeof(unsigned char));
	unsigned char* debug = (unsigned char*)malloc(nrows*ncols*nchans*sizeof(unsigned char));
	unsigned char* d_debug;
	hipMalloc(&d_debug,nchans*nrows*ncols*sizeof(unsigned char));

	// define a shortcut to the host data arrays
	unsigned char* data_l = ((unsigned char*)(im_l.data));
	unsigned char* data_r = ((unsigned char*)(im_r.data));

	// initialize the outputs (otherwise changes persist between runtimes, hard to debug):
	int tpb = 1024;
	int bpg = nrows*ncols*sizeof(unsigned char) / tpb + 1;
	gpu_memset<<<bpg, tpb>>>(d_out,25,nrows*ncols*sizeof(unsigned char));
	// gpu_perror("memset1");
	gpu_memset<<<nchans*bpg, tpb>>>(d_debug,25,nchans*nrows*ncols*sizeof(unsigned char));
	// gpu_perror("memset2");

	// check some values before calling the asw_kernel
	size_t reference_window_size = (2*win_rad+BLOCK_SIZE)*(2*win_rad+BLOCK_SIZE)*sizeof(unsigned char)*nchans;
	size_t target_window_size = (2*win_rad+ndisp+BLOCK_SIZE)*(BLOCK_SIZE+2*win_rad)*sizeof(unsigned char)*nchans;
	size_t shared_size = target_window_size+reference_window_size;

	if(shared_size > 47000){
		printf("FATAL ERROR: shared_size for asw_kernel exceeds the device limit (48 kB), exiting\n");
		return 1;
	}
	
	//copy the host input data to the device
    hipMemcpy(d_im_l, data_l, nchans*nrows*ncols*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_im_r, data_r, nchans*nrows*ncols*sizeof(unsigned char), hipMemcpyHostToDevice);

	// start the timer	
	check_timer(NULL, &ts);
	// call the asw_kernel
	dim3 blocksPerGrid(22,21);
	dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE);
	// __global__ void asw_kernel(unsigned char* global_left, unsigned char* global_right, unsigned char* output, unsigned char* debug,
	//		int nrows, int ncols, int nchans, int ndisp, int win_size, int win_rad, float s_sigma, float c_sigma)
    asw_kernel<<<blocksPerGrid, threadsPerBlock, shared_size>>>(d_im_l, d_im_r, d_out, d_debug,
    	nrows, ncols, nchans, ndisp, win_size, win_rad, s_sigma, c_sigma);
    hipDeviceSynchronize();
    check_timer("gpu_asw", &ts);
	// gpu_perror("asw_kernel");

	// copy the device output data to the host
    hipMemcpy(out, d_out, nrows*ncols*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(debug, d_debug, nrows*ncols*nchans*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // make an image and view it:
    cv::Mat im_out(nrows,ncols,CV_8UC1,out);
    cv::Mat im_debug(nrows,ncols,CV_8UC3,debug);
	cv::imwrite("out/gpu_asw.png",im_out*255/ndisp);
	//cv::imshow("window",im_out*255/ndisp);
	//cv::waitKey(0);

	// cleanup memory
	hipFree(d_im_l);
	hipFree(d_im_r);
	hipFree(d_out);
	hipFree(d_debug);
	free(out);
	free(debug);

	return 0;
}

int main(int argc, char** argv){
	// spacial and color sigmas
	int s_sigma = 5;
	int c_sigma = 50;
	// number of disparities to check
	int ndisp = 64;
	// input images
	cv::Mat im_l = cv::imread("l.png");
	cv::Mat im_r = cv::imread("r.png");

	return asw(im_l, im_r, ndisp, s_sigma, c_sigma);
}
